#include "hip/hip_runtime.h"
#define N 2
# define TAU 0.0001
#define T0 0
//#define TF 0.034492738
#define TF 12.58984937
#include "structs.h"
#include<iostream>
#include <cmath>
#include <fstream>
using namespace std;



//variables - это массив в котором хранится только s и v/
 __device__ double*  NewtonEq(double *variables, double t,double f, double mass) {
        double *res = (double *)malloc(2 * sizeof(double));
        res[0] = variables[1];
        res[1]=f/mass;
        return res;
    }

__device__ double* calculateNextValue(double tau, double *variables,  double t, double mass, double f){
        // vector<double> nVariables(variables.size());
        int size = 2;
        double *nVariables = (double *)malloc(size * sizeof(double));
        double *variablesK2 = (double *)malloc(size * sizeof(double));
        double *variablesK3 = (double *)malloc(2 * sizeof(double));
        double *variablesK4 = (double *)malloc(2 * sizeof(double));
        double *K = (double *)malloc(2 * sizeof(double));
                
        double* k1=NewtonEq(variables,t,f,mass);

        for(int k =0;k<size;k++){
            variablesK2[k]=variables[k]+tau/2 * k1[k];
        }
        double* k2=NewtonEq(variablesK2,t+tau/2,f ,mass);

        for(int k =0;k<size;k++){
            variablesK3[k]=variables[k]+tau/2 * k2[k];
        }
        double* k3 = k3=NewtonEq(variablesK3,t+tau/2,f,mass);

        for(int k =0;k<size;k++){
            variablesK4[k]=variables[k]+tau * k3[k];
        }
        double* k4=NewtonEq(variablesK4,t+tau, f,mass);

        for (int l = 0; l < size; ++l) {
            K[l]=1.0/6.0 * (k1[l]+2*k2[l]+2*k3[l]+k4[l]);
        }

        for (int j = 0; j < size; j++){
            nVariables[j] =variables[j]+tau*K[j]; // сейчас кол-во элементов равно i
        }
        free(variablesK2);free(variablesK3);free(variablesK4);
        free(K);
        free(k1);free(k2);free(k3);free(k4);
        return nVariables;

    }

__device__ void calcAndSetSpeedAndCoord(Particle particle,double tau,double t){
        double * variables = (double*)malloc(2 * sizeof(double));
        variables[0]=particle.s->x;
        variables[1]=particle.v->x;
        double* nVariables = calculateNextValue(tau,variables,t, particle.mass , particle.f->x);
        particle.s->x = nVariables[0];
        particle.v->x = nVariables[1];


        variables[0]=particle.s->y;
        variables[1]=particle.v->y;
        nVariables = calculateNextValue(tau,variables,t, particle.mass , particle.f->y);
        particle.s->y = nVariables[0];
        particle.v->y = nVariables[1];
    }

__global__ void calculate(Particle *particles, double* tau, double* t) {
        const double G = 498217402368e-12;
        int i = blockIdx.x;
        Point *f = (struct Point*)malloc (sizeof (struct Point));
        Particle pi = particles[i];
        for (int j = 0; j < N; j++) {
            if (i == j) continue;

            Particle pj = particles[j];
            double dx = pj.s->x - pi.s->x;
            double dy = pj.s->y - pi.s->y;
            double r = sqrt(dx*dx + dy*dy);
            double Gmij =G * pi.mass * pj.mass;
            f->x += Gmij*dx /(r*r*r);
            f->y += Gmij*dy /(r*r*r);
        }
        particles[i].f=f;
        particles[i].f->x +=100;
        calcAndSetSpeedAndCoord(particles[i],*tau,*t);
        //free(f);
}

// void writingDataInFile(double x, double y , int k){ // k - номер тела
//     const string pathToFile = "/home/andrey/CLionProjects/gravitationalProblem/cuda/files/"+to_string(k)+".txt";
//     ofstream file(pathToFile, ios_base::app);
//     file<<x<<" "<<y<<endl;
//     file.close();
// }

__global__ void add(int *a, int *b, int *c){
    printf("Hello, world from the device!\n");
    *c= *a + *b;
}

// int main(){
//     // cleanFiles(N);
//     srand (time(NULL));
//     //auto particles = getInitRandom();
//     Particle* particles = getInitFor2Particle();
//     double t = T0;
//     double tau=TAU;
//     double tf=TF;
//     double *dev_tau;
//     double *dev_T;
//     Particle* dev_particles;
//     int countIter = (tf-t)/tau;
    
//     int size= N * sizeof(Particle);
//     hipMalloc((void**)&dev_particles, size);
//     hipMalloc((void**)&dev_tau, sizeof(double));
//     hipMalloc((void**)&dev_T, sizeof(double));
    
//     hipMemcpy(dev_particles, particles, size, hipMemcpyHostToDevice);
//     hipMemcpy(dev_tau, &tau, sizeof(double), hipMemcpyHostToDevice);
        
//     cout <<particles[0].f->x << '\t'<<particles[0].s->y <<endl;
//     for (int i = 0; i < countIter; i++) {
//         hipMemcpy(dev_T, &t, sizeof(double), hipMemcpyHostToDevice);
//         cout<<"T="<<*dev_T<<'\t'<<t<<endl;
//      //   *dev_T+=0.01;
//         calculate<<<N,1>>>(dev_particles,dev_tau,dev_T);
//         t+=tau;
//         hipMemcpy(particles, dev_particles, size, hipMemcpyDeviceToHost);
//         cout <<particles[0].f->x << '\t'<<particles[0].s->y <<endl;
//         // writingDataInFile(particles[0].s->x,particles[0].s->y,0);
//         // writingDataInFile(particles[1].s->x,particles[1].s->y,1);     
//     }

//     free(particles);
//     hipFree(dev_particles);hipFree(dev_T);hipFree(dev_tau);
//     return 0;
// }

int main(void){
    int a, b, c;
    int *dev_a, *dev_b, *dev_c;
    int size= sizeof(int);
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    a=3;
    b=1;
    hipMemcpy(dev_a,&a,size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b,&b,size, hipMemcpyHostToDevice);

    add<<< 1,1 >>>(dev_a,dev_b,dev_c);
    hipDeviceSynchronize();
    hipMemcpy(&c,dev_c,size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    cout<<c<<endl;
    return 0;
}

