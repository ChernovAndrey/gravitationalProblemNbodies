#include "hip/hip_runtime.h"

#include<iostream>
using namespace std;
#include <fstream>
#include"structs.h"
#include<string>


__device__ Particle  **vecParticlesK;
__device__ int dM; // кол-во необходимых массивов.

__device__ void  calcForces(Particle *particles, int i, int N,double G){ //i-номер тела
	Point f={0,0};
	Particle pi = particles[i];
	// printf("i = %d ,  N= %d\n",i,N);
	for (int j = 0; j < N; j++) {
	    // printf("%s\n", "start i iteration");
	    if (i == j) continue;

	    Particle pj = particles[j];
	    double dx = pj.s.x - pi.s.x;
	    double dy = pj.s.y - pi.s.y;
	    // printf("dx = %f ,  dy=%f\n",dx,dy);    
	    double r = sqrt(dx*dx + dy*dy);
	    double Gmij =G * pi.mass * pj.mass;
	    f.x += Gmij*dx /(r*r*r);
	    f.y += Gmij*dy /(r*r*r);
	    // printf("f = %f    %f\n",f.x,f.y);
    
	}
	particles[i].f=f;

}

__device__ void copyParticlesToVecParticlesK(Particle * particles, int i){
	for(int j=0;j<dM;j++){
		vecParticlesK[j][i] = particles[i];
	}
}
__device__ void moveParticle(Particle *&particles, Particle *ParticlesK, double step, int i) { // i - номер тела
            particles[i].s.x += step * ParticlesK[i].v.x;//подставляем непосредственно уравнение Ньютона.
            particles[i].s.y += step * ParticlesK[i].v.y;//подставляем непосредственно уравнение Ньютона.

            particles[i].v.x += step * ParticlesK[i].f.x /
                     ParticlesK[i].mass;//подставляем непосредственно уравнение Ньютона.
            particles[i].v.y += step * ParticlesK[i].f.y /
                    ParticlesK[i].mass;//подставляем непосредственно уравнение Ньютона.
       
}

// __device__ void moveParticle(Particle *&particles, double step[], int i) { // i - номер тела
// 		for (int j = 0; j < vecParticlesK.size(); ++j) {
//             particles[i].s.x += step[j] * vecParticlesK[j][i]->v.x;//подставляем непосредственно уравнение Ньютона.
//             particles[i].s.y += step[j] * vecParticlesK[j][i]->v.y;//подставляем непосредственно уравнение Ньютона.

//             particles[i].v.x += step[j] * vecParticlesK[j][i]->f.x /
//                     vecParticlesK[j][i]->mass;//подставляем непосредственно уравнение Ньютона.
//             particles[i].v.y += step[j] * vecParticlesK[j][i]->f.y /
//                     vecParticlesK[j][i]->mass;//подставляем непосредственно уравнение Ньютона.
//      	   }
// }


__device__ void moveParticle(Particle *&particles, Particle* vecParticlesK[], double step[], int i, int sizeVecParticles) { // i - номер тела
		for (int j = 0; j < sizeVecParticles; ++j) {
            particles[i].s.x += step[j] * vecParticlesK[j][i].v.x;//подставляем непосредственно уравнение Ньютона.
            particles[i].s.y += step[j] * vecParticlesK[j][i].v.y;//подставляем непосредственно уравнение Ньютона.

            particles[i].v.x += step[j] * vecParticlesK[j][i].f.x /
                    vecParticlesK[j][i].mass;//подставляем непосредственно уравнение Ньютона.
            particles[i].v.y += step[j] * vecParticlesK[j][i].f.y /
                    vecParticlesK[j][i].mass;//подставляем непосредственно уравнение Ньютона.
     	   }
}



__device__ void calculateRK1(Particle *&particles, double tau,int i, int N, double G) { // i - номер тела.

    calcForces(particles,i,N,G);
    __syncthreads();
    moveParticle(particles,particles,tau,i);
}


__device__ void calculateRK2(Particle  *&particles, double tau, int i, int N, double G) {
	calcForces(particles,i,N,G);
	 __syncthreads();
	copyParticlesToVecParticlesK(particles,i);
    __syncthreads();
    moveParticle(vecParticlesK[0],vecParticlesK[0],tau/2.0,i);
    __syncthreads();
    calcForces(vecParticlesK[0],i,N,G);
    __syncthreads();
    moveParticle(particles,vecParticlesK[0],tau,i);
}



__device__ void calculateRK3(Particle *&particles, double tau, int i, int N, double G) {

    calcForces(particles,i,N,G);
	__syncthreads();
	copyParticlesToVecParticlesK(particles,i);
    __syncthreads();
    moveParticle(vecParticlesK[0],vecParticlesK[0],tau/2.0,i);
    __syncthreads();
    calcForces(vecParticlesK[0],i,N,G);
	__syncthreads();
	moveParticle(vecParticlesK[1],(Particle*[]){particles,vecParticlesK[0]},(double[]){-tau,2*tau},i,2);
    __syncthreads();
    calcForces(vecParticlesK[1],i,N,G);
    __syncthreads();
    moveParticle(particles,(Particle*[]){particles,vecParticlesK[0],vecParticlesK[1]},(double[]){tau/6.0,4*tau/6.0,tau/6.0},i,3);

}

__device__ void calculateRK4(Particle *&particles, double tau, int i, int N, double G) {

    calcForces(particles,i,N,G);
    __syncthreads();
   	copyParticlesToVecParticlesK(particles,i);
   	__syncthreads();
    moveParticle(vecParticlesK[0],vecParticlesK[0],tau/2.0,i);
	__syncthreads();
    calcForces(vecParticlesK[0],i,N,G);
	__syncthreads();
	moveParticle(vecParticlesK[1],vecParticlesK[0],tau/2.0,i);
    __syncthreads();
    calcForces(vecParticlesK[1],i,N,G);
	__syncthreads();
	moveParticle(vecParticlesK[2],vecParticlesK[1],tau,i);
    __syncthreads();
    calcForces(vecParticlesK[2],i,N,G);
    __syncthreads();
    moveParticle(particles,(Particle*[]){particles,vecParticlesK[0],vecParticlesK[1],vecParticlesK[2]},(double[]){tau/6.0,2*tau/6.0,2*tau/6.0,tau/6.0},i,4);

}

__global__ void calculate(Particle *particles,double *t, double *tau, int *N){
	int i =  threadIdx.x + blockIdx.x * blockDim.x;;
	// int i = threadIdx.x;
	// printf("blockdim=%d\n",blockDim.x);
	const double G = 498217402368e-12;
	// const double G = 4.0;
	calculateRK1(particles,*tau,i,*N,G);
	__syncthreads();
}



void copyParticlesDeviceToHost(Particle *&particles, Particle *&dev_particles, int size){
	// hipMemcpy(&particles, &dev_particles, size, hipMemcpyDeviceToHost);
	hipMemcpy(&particles[0].s, &dev_particles[0].s, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[0].v, &dev_particles[0].v, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[0].f, &dev_particles[0].f, sizeof(struct Point), hipMemcpyDeviceToHost);
    hipMemcpy(&particles[1].s, &dev_particles[1].s, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[1].v, &dev_particles[1].v, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[1].f, &dev_particles[1].f, sizeof(struct Point), hipMemcpyDeviceToHost);
}

__global__ void createVecParticlesK(int *M, int *N){// M - кол-во необходимых массивов.
	 vecParticlesK = new Particle*[*M];
	 for(int i=0;i<*M;i++){
	 	vecParticlesK[i] = new Particle[*N];
	 }
	 dM = *M;
}


__global__ void deleteVecParticlesK(int *M, int *N){// M - кол-во необходимых массивов.
	 for(int i=0;i<*M;i++){
	 	delete[] vecParticlesK[i];
	 }
	 delete[] vecParticlesK;
}

int main( void ) {
// cleanFiles(N);
srand (time(NULL));

Particle *particles=NULL;
Particle *dev_particles=NULL;
int countBlocks = 13;
int countThreads = 128;
int N=countBlocks*countThreads;
// int N=2;
int M=3;
getInitRandom(particles,dev_particles,N);
// getInitFor2Particle(particles,dev_particles);
double t = 0.0;
// double tau=0.0001;
double tau =0.001;
//double tf = 0.001;
// double tf=12.58984937;
double tf = 0.01;
// double tf=2*M_PI;
double *dev_tau;
double *dev_T;

int *dev_N;
int *dev_M;
int countIter = (tf-t)/tau;
int size= N*sizeof (struct Particle);
//int size= N * sizeof(Particle);
hipMalloc((void**)&dev_tau, sizeof(double));
hipMalloc((void**)&dev_T, sizeof(double));
hipMalloc((void**)&dev_N,sizeof(int));
hipMalloc((void**)&dev_M,sizeof(int));

hipMemcpy(dev_N,&N,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_M,&M,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_tau, &tau, sizeof(double), hipMemcpyHostToDevice);
createVecParticlesK<<<1,1>>>(dev_M,dev_N);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);

cout<<"count iter="<<countIter<<endl;


cout.precision(14);
for (int i = 0; i < countIter; i++) {
	 t+=tau;	
     hipMemcpy(dev_T, &t, sizeof(double), hipMemcpyHostToDevice);
     calculate<<<countBlocks,countThreads>>>(dev_particles,dev_T,dev_tau,dev_N);
     // calculate<<<1,2>>>(dev_particles,dev_T,dev_tau,dev_N);
     copyParticlesDeviceToHost(particles,dev_particles,size);
     // cout << particles[0].s.x << "   "<< particles[0].s.y <<endl;
     // cout << particles[1].s.x << "   "<< particles[1].s.y <<endl;
 }
hipEventRecord(stop);
hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);    
cout<<"time execute:"<<milliseconds<<endl;
hipEventDestroy (start);
hipEventDestroy (stop);
hipFree( dev_particles );
hipFree( dev_tau );
hipFree( dev_T );
free(particles); // все надо освободить
deleteVecParticlesK<<<1,1>>>(dev_M,dev_N);
return 0;
}