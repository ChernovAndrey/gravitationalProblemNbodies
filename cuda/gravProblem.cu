#include "hip/hip_runtime.h"

#include<iostream>
using namespace std;
#include <fstream>
#include"structs.h"
#include<string>
#include <stdio.h>
#include <stdlib.h>
#include <sstream>
__global__ void add( int *a, int *b, int *c ) {
printf("%d\n", *a);
*c = *a + *b;
}

 __device__ double*  NewtonEq(double *variables, double t,double f, double mass) {
        double *res = (double *)malloc(2 * sizeof(double));
        // double res[2];
        res[0] = variables[1];
        res[1]=f/mass;
        return res;
    }

__device__ void calculateNextValue(double tau, double *variables,  double t, double mass, double f){
        // vector<double> nVariables(variables.size());
        const int size = 2;
        // double *variablesK2 = (double *)malloc(size * sizeof(double));
        // double *variablesK3 = (double *)malloc(2 * sizeof(double));
        // double *variablesK4 = (double *)malloc(2 * sizeof(double));
        // double *K = (double *)malloc(2 * sizeof(double));        
        double k1[2];
        double k2[2];
        double k3[2];
        double k4[2];	
        // double* k1=NewtonEq(variables,t,f,mass);
        k1[0]=variables[1];
        k1[1]=f/mass;

        // double nVariables[2];
        double variablesK2[2];
        double variablesK3[2];
        double variablesK4[2];
        double K[2];
        for(int k =0;k<size;k++){
            variablesK2[k]=variables[k]+tau/2 * k1[k];
        }
        // double* k2=NewtonEq(variablesK2,t+tau/2,f ,mass);
        k2[0]=variablesK2[1];
        k2[1]=k1[1];

        for(int k =0;k<size;k++){
            variablesK3[k]=variables[k]+tau/2 * k2[k];
        }
        // double* k3 = k3=NewtonEq(variablesK3,t+tau/2,f,mass);
        k3[0]=variablesK3[1];
        k3[1]=k1[1];
        for(int k =0;k<size;k++){
            variablesK4[k]=variables[k]+tau * k3[k];
        }
        // double* k4=NewtonEq(variablesK4,t+tau, f,mass);
        k4[0]=variablesK4[1];
        k4[1]=k1[1];
        for (int l = 0; l < size; ++l) {
            K[l]=1.0/6.0 * (k1[l]+2*k2[l]+2*k3[l]+k4[l]);
        }

        for (int j = 0; j < size; j++){
            variables[j] =variables[j]+tau*K[j]; // сейчас кол-во элементов равно i
        }
    //новые значения в variables
    }

__device__ void calcAndSetSpeedAndCoord(Particle &particle,double tau,double t){
        //double * variables = (double*)malloc(2 * sizeof(double));
		double variables[2];        
        variables[0]=particle.s.x;
        variables[1]=particle.v.x;
        calculateNextValue(tau,variables,t, particle.mass , particle.f.x);
        particle.s.x = variables[0];
        particle.v.x = variables[1];


        variables[0]=particle.s.y;
        variables[1]=particle.v.y;
        calculateNextValue(tau,variables,t, particle.mass , particle.f.y);
        particle.s.y = variables[0];
        particle.v.y = variables[1];
        // delete nVariables;
    }

// string convertIntToString(int a){
// 	// char *intStr = itoa(a);
// 	// return  string(intStr);

// 	stringstream ss;
//  	ss << a;
// 	return ss.str();
// }

// void writingDataInFile(double x, double y , int k){ // k - номер тела
//     string pathToFile = "files/"+convertIntToString(k)+".txt";
//     char *cpath = new char[pathToFile.length() + 1];
// 	strcpy(cpath, pathToFile.c_str());
// 	// do stuff
//     FILE* f = fopen(cpath, "a");
// 	fprintf(f, "%f %f", x,y);
// 	fclose(f);
// 	delete [] cpath;
//     //ofstream file(pathToFile, ios_base::app);
//     // file<<x<<" "<<y<<endl;
//     // file.close();
// }



__device__ void  calcForces(Particle *particles, int i, double G, int *N){
	Point f;
	Particle pi = particles[i];
	for (int j = 0; j < *N; j++) {
	    // printf("%s\n", "start i iteration");
	    if (i == j) continue;

	    Particle pj = particles[j];
	    double dx = pj.s.x - pi.s.x;
	    double dy = pj.s.y - pi.s.y;
	    double r = sqrt(dx*dx + dy*dy);
	    double Gmij =G * pi.mass * pj.mass;
	    f.x += Gmij*dx /(r*r*r);
	    f.y += Gmij*dy /(r*r*r);
	}
	particles[i].f=f;

}

__global__ void calculate(Particle *particles,double *t, double *tau, int *N){
	int i =  threadIdx.x + blockIdx.x * blockDim.x;;
	// printf("blockdim=%d\n",blockDim.x);
	const double G = 498217402368e-12;
	// printf("N=%d\n", *N);
	// printf("%s\n","start calculate" );
    // printf("in gpu start %f  %f\n",particles[i].s.x,particles[i].s.y);
	// Point f;
	// Particle pi = particles[i];
	// for (int j = 0; j < *N; j++) {
	//     // printf("%s\n", "start i iteration");
	//     if (i == j) continue;

	//     Particle pj = particles[j];
	//     double dx = pj.s.x - pi.s.x;
	//     double dy = pj.s.y - pi.s.y;
	//     double r = sqrt(dx*dx + dy*dy);
	//     double Gmij =G * pi.mass * pj.mass;
	//     f.x += Gmij*dx /(r*r*r);
	//     f.y += Gmij*dy /(r*r*r);
	// }
	// particles[i].f=f;
	 calcForces(particles,i,G,N);
	__syncthreads();
	calcAndSetSpeedAndCoord(particles[i],*tau,*t);
	// printf("f=%f %f\n",f.x,f.y);
	__syncthreads();
}

__global__ void printParticles(Particle *particles){
	printf("sx = %f\n", particles[0].s.x);
}

void copyParticlesDeviceToHost(Particle *&particles, Particle *&dev_particles, int size){
	// hipMemcpy(&particles, &dev_particles, size, hipMemcpyDeviceToHost);
	hipMemcpy(&particles[0].s, &dev_particles[0].s, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[0].v, &dev_particles[0].v, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[0].f, &dev_particles[0].f, sizeof(struct Point), hipMemcpyDeviceToHost);
    hipMemcpy(&particles[1].s, &dev_particles[1].s, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[1].v, &dev_particles[1].v, sizeof(struct Point), hipMemcpyDeviceToHost);
    // hipMemcpy(&particles[1].f, &dev_particles[1].f, sizeof(struct Point), hipMemcpyDeviceToHost);
}


int main( void ) {
// cleanFiles(N);
srand (time(NULL));

Particle *particles=NULL;
Particle *dev_particles=NULL;
int countBlocks = 13;
int countThreads = 64;
int N=countBlocks*countThreads;
getInitRandom(particles,dev_particles,N);
// getInitFor2Particle(particles,dev_particles);
double t = 0.0;
// double tau=0.0001;
double tau =1.0;
//double tf = 0.001;
// double tf=12.58984937;
double tf=10.0;
double *dev_tau;
double *dev_T;

int *dev_N;
int countIter = (tf-t)/tau;
int size= N*sizeof (struct Particle);
//int size= N * sizeof(Particle);
hipMalloc((void**)&dev_tau, sizeof(double));
hipMalloc((void**)&dev_T, sizeof(double));
hipMalloc((void**)&dev_N,sizeof(double));
   
hipMemcpy(dev_N,&N,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_tau, &tau, sizeof(double), hipMemcpyHostToDevice);
// hipMemcpy(dev_T, &t, sizeof(double), hipMemcpyHostToDevice);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);

cout<<"count iter="<<countIter<<endl;
for (int i = 0; i < countIter; i++) {
  //   cout<<"start iter in host"<<endl;
     hipMemcpy(dev_T, &t, sizeof(double), hipMemcpyHostToDevice);
     // cout<<"T="<<*dev_T<<'\t'<<t<<endl;
  //   *dev_T+=0.01;
     calculate<<<countBlocks,countThreads>>>(dev_particles,dev_T,dev_tau,dev_N);
     t+=tau;
     copyParticlesDeviceToHost(particles,dev_particles,size);
     // cout <<particles[0].s.x << " "<<particles[0].s.y <<endl;
     // cout <<particles[1].s.x << " "<<particles[1].s.y <<endl;
     // cout<<"finish iter="<<i<<endl;
      // writingDataInFile(particles[0].s.x,particles[0].s.y,0);
      // writingDataInFile(particles[1].s.x,particles[1].s.y,1);
      // if (i%1000==0){
      // 	cout<<"finish iter="<<i<<endl;
      // }     
 }
hipEventRecord(stop);
hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);    
cout<<"time execute:"<<milliseconds<<endl;
hipEventDestroy (start);
hipEventDestroy (stop);
hipFree( dev_particles );
hipFree( dev_tau );
hipFree( dev_T );
free(particles); // все надо освободить
return 0;
}